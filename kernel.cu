#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>

hipError_t findPIInitializer(int *circle_points, int *square_points, int size, int const INTERVALO);

__global__ void findPIKernel(int *d_circle_points,  int *d_square_points, int const INTERVALO, unsigned int seed)
{
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);

    /* hiprand works like rand - except that it takes a state as a parameter */
    double rand_x, rand_y, origin_dist;

    rand_x = double(hiprand(&state) % INTERVALO) / INTERVALO;
    rand_y = double(hiprand(&state) % INTERVALO) / INTERVALO;

    // Distance between (x, y) from the origin 
    origin_dist = rand_x * rand_x + rand_y * rand_y;

    // Checking if (x, y) lies inside the define 
    // circle with R=1 
    if (origin_dist <= 1)
        circle_points+=1;

    // Total number of points generated 
    square_points+=1;
}

int main()
{
   

    int circle_points, square_points; // copias locais do numero de pontos dentro do circulo e fora(no quadrado como um todo)
    int *d_circle_points, *d_square_points; // device copies of a, b, c
    int size = sizeof(int);
    int const INTERVALO = 10;

    // Add vectors in parallel.
    hipError_t cudaStatus = findPIInitializer(circle_points, square_points, size, INTERVALO);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t findPIInitializer(int *circle_points, int *square_points, int size, int const INTERVALO)
{
    int *d_circle_points, *d_square_points;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate space for device copies of a, b, c.
    cudaStatus = hipMalloc((void **)&d_circle_points, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&d_square_points, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_circle_points, circle_points, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(d_square_points, square_points, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    srand(time(NULL));
    int threadsPerBlock = 256;
    int blocksPerGrid = (INTERVALO + threadsPerBlock - 1) / threadsPerBlock;
    findPIKernel<<<blocksPerGrid, threadsPerBlock>>>(d_circle_points, d_square_points, INTERVALO, time(NULL));

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "findPIKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(circle_points d_circle_points, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(square_points d_square_points, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
